#include "hip/hip_runtime.h"
#include <ml/conv2d/gpu.h>

#define ENABLE_DEVICE_FUNCTIONS
#include "../common_nn.ipp"


/*
 * Note: CNNs has a concept of "kernel" and CUDA has a concept of "kernel". That could
 * cause confusion. In this file we will only talk about CNN kernels, and we will avoid
 * using the term "kernel" to talk about that CUDA concept--we will use alternate words
 * for that instead.
 */


namespace ml
{
namespace conv2d
{
namespace gpu
{


/*
 * A block is filled by threads with access to shared memory.
 * This way we are using the most threads allowed: 1024
 */
#define BLOCK_SIZE_Y 32
#define BLOCK_SIZE_X 32


template <u32 INPUT_COMPONENTS, u32 KERNEL_ROWS, u32 KERNEL_COLS, u32 NUM_KERNELS>
__global__
void
__launch_bounds__(BLOCK_SIZE_Y*BLOCK_SIZE_X, 1)
gpu_conv2d_multi_input(
        const fml* inputPtr,  u32 inputRows,   u32 inputCols,
        const fml* kernelPtr, u32 kernelStepY, u32 kernelStepX,
        const fml* kernelBiases, fml scaleFactor,
              fml* outputPtr, u32 outputRows, u32 outputCols)
{
    // We use shared memory so that each global memory value only must be read once!
    // Makes everything much much faster.
    // We will keep only one component of the input block in shared memory at a time.
    // We will keep all the components of all the kernels in shared memory at the same time though!
    extern __shared__ fml memory_shared[];
    fml* input_shared  = memory_shared;
    fml* kernel_shared = input_shared  + BLOCK_SIZE_Y * BLOCK_SIZE_X;
    fml* bias_shared   = kernel_shared + KERNEL_ROWS  * KERNEL_COLS   * INPUT_COMPONENTS * NUM_KERNELS;

    // Useful things to have:
    inputPtr  += blockIdx.z * inputRows * inputCols * INPUT_COMPONENTS;
    outputPtr += blockIdx.z * outputRows * outputCols * NUM_KERNELS;
    i32 global_y = blockIdx.y * (BLOCK_SIZE_Y-KERNEL_ROWS+1) + threadIdx.y;  global_y -= KERNEL_ROWS/2;
    i32 global_x = blockIdx.x * (BLOCK_SIZE_X-KERNEL_COLS+1) + threadIdx.x;  global_x -= KERNEL_COLS/2;

    // All threads will help copy values into the shared memory. But not
    // all threads will be required to calculate output values. Only
    // threads that have all the following attributes will be required
    // to calculate output values:
    //   - be inside the effective block,
    //   - be inside the input, and
    //   - be aligned to the kernel step size.
    bool isInsideEffectiveBlock =
                (threadIdx.y >= KERNEL_ROWS/2 && (threadIdx.y - KERNEL_ROWS/2) < (BLOCK_SIZE_Y-KERNEL_ROWS+1) &&
                 threadIdx.x >= KERNEL_COLS/2 && (threadIdx.x - KERNEL_COLS/2) < (BLOCK_SIZE_X-KERNEL_COLS+1));
    bool isInsideInput =
                (global_y >= 0 && global_y < inputRows &&
                 global_x >= 0 && global_x < inputCols);
    bool isAlignedToKerenlStep =
                ((global_y % kernelStepY) == 0 &&
                 (global_x % kernelStepX) == 0);
    bool isOutputThread =
                (isInsideEffectiveBlock &&
                 isInsideInput &&
                 isAlignedToKerenlStep);

    // Copy all the kernels into shared memory.
    {
        u32 sizeToCopy = KERNEL_ROWS  * KERNEL_COLS   * INPUT_COMPONENTS * NUM_KERNELS;
        for (u32 copyIndex = threadIdx.y * BLOCK_SIZE_X + threadIdx.x; copyIndex < sizeToCopy; copyIndex += BLOCK_SIZE_Y * BLOCK_SIZE_X)
        {
            kernel_shared[copyIndex] = kernelPtr[copyIndex];
        }
        sizeToCopy = NUM_KERNELS;
        for (u32 copyIndex = threadIdx.y * BLOCK_SIZE_X + threadIdx.x; copyIndex < sizeToCopy; copyIndex += BLOCK_SIZE_Y * BLOCK_SIZE_X)
        {
            bias_shared[copyIndex] = kernelBiases[copyIndex];
        }
    }

    // For each component of the input, we will process it independently.
    fml accumulators[NUM_KERNELS];
    for (u32 inputComponentIndex = 0; inputComponentIndex < INPUT_COMPONENTS; inputComponentIndex++)
    {
        // Copy this channel into the shared memory.
        if (isInsideInput)
        {
            input_shared[threadIdx.y * BLOCK_SIZE_X + threadIdx.x] = *(inputPtr + global_y * inputCols * INPUT_COMPONENTS + global_x * INPUT_COMPONENTS + inputComponentIndex);
        }
        else
        {
            input_shared[threadIdx.y * BLOCK_SIZE_X + threadIdx.x] = FML(0.0);
        }

        // Don't move on until all threads have copied the values they are each responsible for.
        // Because we are about to use all these values in a calculation.
        __syncthreads();

        // Do the convolution of this channel, and add it to the accumulator.
        // Not all threads have work here, because some threads exist only to copy the apron
        // values into shared memory, and some threads are not aligned to the kernel step size.
        if (isOutputThread)
        {
            const fml* input_start = input_shared + (threadIdx.y - KERNEL_ROWS/2) * BLOCK_SIZE_X + threadIdx.x - KERNEL_COLS/2;

            for (u32 kernelIndex = 0; kernelIndex < NUM_KERNELS; kernelIndex++)
            {
                // The calculation.
                const fml* kernel_start = kernel_shared + kernelIndex * KERNEL_ROWS * KERNEL_COLS * INPUT_COMPONENTS + inputComponentIndex;
                fml result = FML(0.0);
                for (u32 kernelRowIndex = 0; kernelRowIndex < KERNEL_ROWS; kernelRowIndex++)
                {
                    const fml* kernel_row = kernel_start + kernelRowIndex * KERNEL_COLS * INPUT_COMPONENTS;
                    const fml* input_row = input_start + kernelRowIndex * BLOCK_SIZE_X;
                    for (u32 kernelColIndex = 0; kernelColIndex < KERNEL_COLS; kernelColIndex++)
                    {
                        result += kernel_row[kernelColIndex * INPUT_COMPONENTS]
                                * input_row[kernelColIndex];
                    }
                }

                // The storage to the accumulator.
                if (inputComponentIndex == 0)
                {
                    accumulators[kernelIndex] = (result + bias_shared[kernelIndex]) * scaleFactor;
                }
                else
                {
                    accumulators[kernelIndex] += result * scaleFactor;
                }
            }
        }

        // Don't loop back up and start messing with shared memory again until all threads are finished
        // with the calculation above (which uses the current shared memory values).
        __syncthreads();
    }

    // Output the final results.
    if (isOutputThread)
    {
        outputPtr += global_y/kernelStepY * outputCols * NUM_KERNELS + global_x/kernelStepX * NUM_KERNELS;
        for (u32 kernelIndex = 0; kernelIndex < NUM_KERNELS; kernelIndex++)
        {
            outputPtr[kernelIndex] = accumulators[kernelIndex];
        }
    }
}


#define SWITCH_KERNEL_DIMS(inputComponents, numKernels) \
    switch ((kernelRows * 0x10) + kernelCols) \
    { \
        case 0x33: \
            gpu_conv2d_multi_input<inputComponents, 3, 3, numKernels><<<gridSize, blockSize, sharedMemNeeded>>>( \
                inputPtr,  inputRows,   inputCols, \
                kernelPtr, kernelStepY, kernelStepX, \
                kernelBiases, scaleFactor, \
                outputPtr, outputRows, outputCols); \
            break; \
 \
        case 0x35: \
            gpu_conv2d_multi_input<inputComponents, 3, 5, numKernels><<<gridSize, blockSize, sharedMemNeeded>>>( \
                inputPtr,  inputRows,   inputCols, \
                kernelPtr, kernelStepY, kernelStepX, \
                kernelBiases, scaleFactor, \
                outputPtr, outputRows, outputCols); \
            break; \
 \
        case 0x37: \
            gpu_conv2d_multi_input<inputComponents, 3, 7, numKernels><<<gridSize, blockSize, sharedMemNeeded>>>( \
                inputPtr,  inputRows,   inputCols, \
                kernelPtr, kernelStepY, kernelStepX, \
                kernelBiases, scaleFactor, \
                outputPtr, outputRows, outputCols); \
            break; \
 \
        case 0x53: \
            gpu_conv2d_multi_input<inputComponents, 5, 3, numKernels><<<gridSize, blockSize, sharedMemNeeded>>>( \
                inputPtr,  inputRows,   inputCols, \
                kernelPtr, kernelStepY, kernelStepX, \
                kernelBiases, scaleFactor, \
                outputPtr, outputRows, outputCols); \
            break; \
 \
        case 0x55: \
            gpu_conv2d_multi_input<inputComponents, 5, 5, numKernels><<<gridSize, blockSize, sharedMemNeeded>>>( \
                inputPtr,  inputRows,   inputCols, \
                kernelPtr, kernelStepY, kernelStepX, \
                kernelBiases, scaleFactor, \
                outputPtr, outputRows, outputCols); \
            break; \
 \
        case 0x57: \
            gpu_conv2d_multi_input<inputComponents, 5, 7, numKernels><<<gridSize, blockSize, sharedMemNeeded>>>( \
                inputPtr,  inputRows,   inputCols, \
                kernelPtr, kernelStepY, kernelStepX, \
                kernelBiases, scaleFactor, \
                outputPtr, outputRows, outputCols); \
            break; \
 \
        case 0x73: \
            gpu_conv2d_multi_input<inputComponents, 7, 3, numKernels><<<gridSize, blockSize, sharedMemNeeded>>>( \
                inputPtr,  inputRows,   inputCols, \
                kernelPtr, kernelStepY, kernelStepX, \
                kernelBiases, scaleFactor, \
                outputPtr, outputRows, outputCols); \
            break; \
 \
        case 0x75: \
            gpu_conv2d_multi_input<inputComponents, 7, 5, numKernels><<<gridSize, blockSize, sharedMemNeeded>>>( \
                inputPtr,  inputRows,   inputCols, \
                kernelPtr, kernelStepY, kernelStepX, \
                kernelBiases, scaleFactor, \
                outputPtr, outputRows, outputCols); \
            break; \
 \
        case 0x77: \
            gpu_conv2d_multi_input<inputComponents, 7, 7, numKernels><<<gridSize, blockSize, sharedMemNeeded>>>( \
                inputPtr,  inputRows,   inputCols, \
                kernelPtr, kernelStepY, kernelStepX, \
                kernelBiases, scaleFactor, \
                outputPtr, outputRows, outputCols); \
            break; \
 \
        default: \
            throw eImpossiblePath(); \
    } \


#define SWITCH_NUM_KERNELS(inputComponents) \
    switch (numKernels) \
    { \
        case 1: SWITCH_KERNEL_DIMS(inputComponents, 1) break; \
        case 2: SWITCH_KERNEL_DIMS(inputComponents, 2) break; \
        case 3: SWITCH_KERNEL_DIMS(inputComponents, 3) break; \
        case 4: SWITCH_KERNEL_DIMS(inputComponents, 4) break; \
        case 5: SWITCH_KERNEL_DIMS(inputComponents, 5) break; \
        case 6: SWITCH_KERNEL_DIMS(inputComponents, 6) break; \
        case 7: SWITCH_KERNEL_DIMS(inputComponents, 7) break; \
        case 8: SWITCH_KERNEL_DIMS(inputComponents, 8) break; \
        case 9: SWITCH_KERNEL_DIMS(inputComponents, 9) break; \
        case 10: SWITCH_KERNEL_DIMS(inputComponents, 10) break; \
        case 11: SWITCH_KERNEL_DIMS(inputComponents, 11) break; \
        case 12: SWITCH_KERNEL_DIMS(inputComponents, 12) break; \
        case 13: SWITCH_KERNEL_DIMS(inputComponents, 13) break; \
        case 14: SWITCH_KERNEL_DIMS(inputComponents, 14) break; \
        case 15: SWITCH_KERNEL_DIMS(inputComponents, 15) break; \
        case 16: SWITCH_KERNEL_DIMS(inputComponents, 16) break; \
        default: \
            throw eInvalidArgument("Unsupported numKernels"); \
    } \


void conv2d_multi_input(
        u32 inputCount,  u32 inputStride,  u32 outputStride,
        const fml* inputPtr,  u32 inputRows,   u32 inputCols,   u32 inputComponents,
        const fml* kernelPtr, u32 kernelRows,  u32 kernelCols,
                              u32 kernelStepY, u32 kernelStepX,
                              u32 numKernels,
        const fml* kernelBiases, fml scaleFactor,
              fml* outputPtr)
{
    assert(inputPtr && inputRows > 0 && inputCols > 0 && inputComponents > 0);
    assert(kernelPtr && (kernelRows % 2) == 1 && (kernelCols % 2) == 1);
    assert(kernelStepY > 0 && kernelStepX > 0 && numKernels > 0);
    assert(kernelBiases);
    assert(outputPtr);

    u32 kernelRadiusY = kernelRows / 2;
    u32 kernelRadiusX = kernelCols / 2;

    dim3 blockSize;
    blockSize.x = BLOCK_SIZE_X;
    blockSize.y = BLOCK_SIZE_Y;
    blockSize.z = 1;

    u32 effectiveBlockSizeY = BLOCK_SIZE_Y - 2*kernelRadiusY;  // Each block of threads will fill
    u32 effectiveBlockSizeX = BLOCK_SIZE_X - 2*kernelRadiusX;  // a smaller block of output, because we need
                                                               // an "apron" so that our kernel doesn't fall of
                                                               // the side and into no-where-land.

    dim3 gridSize;
    gridSize.x = (inputCols-1) / effectiveBlockSizeX + 1;
    gridSize.y = (inputRows-1) / effectiveBlockSizeY + 1;
    gridSize.z = inputCount;

    u32 outputRows = (inputRows - 1) / kernelStepY + 1;
    u32 outputCols = (inputCols - 1) / kernelStepX + 1;

    if (kernelRows != 3 && kernelRows != 5 && kernelRows != 7)
        throw eInvalidArgument("Unsupported kernelRows: must be 3, 5, or 7.");
    if (kernelCols != 3 && kernelCols != 5 && kernelCols != 7)
        throw eInvalidArgument("Unsupported kernelCols: must be 3, 5, or 7.");

    u32 sharedMemNeeded = (BLOCK_SIZE_Y * BLOCK_SIZE_X + kernelRows * kernelCols * inputComponents * numKernels + numKernels) * sizeof(fml);

    switch (inputComponents)
    {
        case 1: SWITCH_NUM_KERNELS(1) break;
//      case 2: SWITCH_NUM_KERNELS(2) break;
//      case 3: SWITCH_NUM_KERNELS(3) break;
//      case 4: SWITCH_NUM_KERNELS(4) break;
//      case 5: SWITCH_NUM_KERNELS(5) break;
        case 6: SWITCH_NUM_KERNELS(6) break;
//      case 7: SWITCH_NUM_KERNELS(7) break;
//      case 8: SWITCH_NUM_KERNELS(8) break;
//      case 9: SWITCH_NUM_KERNELS(9) break;
//      case 10: SWITCH_NUM_KERNELS(10) break;
//      case 11: SWITCH_NUM_KERNELS(11) break;
//      case 12: SWITCH_NUM_KERNELS(12) break;
//      case 13: SWITCH_NUM_KERNELS(13) break;
//      case 14: SWITCH_NUM_KERNELS(14) break;
//      case 15: SWITCH_NUM_KERNELS(15) break;
//      case 16: SWITCH_NUM_KERNELS(16) break;
        default:
            throw eInvalidArgument("Unsupported inputComponents");
    }

    hipError_t errSync  = hipGetLastError();
    if (errSync != hipSuccess)
        throw eRuntimeError(std::string("CUDA launch error: ") + hipGetErrorString(errSync));
}


void conv2d_backprop_multi_input(
        u32 inputCount,  u32 inputStride,  u32 outputStride,
              fml* di_ptr,    u32 inputRows,   u32 inputCols,   u32 inputComponents,
        const fml* kernelPtr, u32 kernelRows,  u32 kernelCols,
                              u32 kernelStepY, u32 kernelStepX,
                              u32 numKernels,
        const fml* kernelBiases, fml scaleFactor,
        const fml* dA_ptr)
{
    // TODO
}


void conv2d_accumError_multi_input(
        u32 inputCount,  u32 inputStride,  u32 outputStride,
        const fml* inputPtr, u32 inputRows,   u32 inputCols,   u32 inputComponents,
              fml* dk_ptr,   u32 kernelRows,  u32 kernelCols,
                             u32 kernelStepY, u32 kernelStepX,
                             u32 numKernels,
              fml* db_ptr, fml scaleFactor,
        const fml* dA_ptr)
{
    // TODO
    // Don't forget to set dk_ptr and db_ptr vectors to zero before you begin.
}


void conv2d_multi_input_with_memcpy(
        u32 inputCount,  u32 inputStride,  u32 outputStride,
        const fml* inputPtr,  u32 inputRows,   u32 inputCols,   u32 inputComponents,
        const fml* kernelPtr, u32 kernelRows,  u32 kernelCols,
                              u32 kernelStepY, u32 kernelStepX,
                              u32 numKernels,
        const fml* kernelBiases, fml scaleFactor,
              fml* outputPtr)
{
    fml* inputPtr_gpu = s_cudaMalloc(inputCount * inputStride);
    fml* kernelPtr_gpu = s_cudaMalloc(kernelRows * kernelCols * inputComponents * numKernels);
    fml* kernelBiases_gpu = s_cudaMalloc(numKernels);
    fml* outputPtr_gpu = s_cudaMalloc(inputCount * outputStride);

    s_cudaCopyHostToDevice(inputPtr_gpu, inputPtr, inputCount * inputStride);
    s_cudaCopyHostToDevice(kernelPtr_gpu, kernelPtr, kernelRows * kernelCols * inputComponents * numKernels);
    s_cudaCopyHostToDevice(kernelBiases_gpu, kernelBiases, numKernels);

    conv2d_multi_input(
        inputCount,  inputStride,  outputStride,
        inputPtr_gpu,  inputRows,   inputCols,   inputComponents,
        kernelPtr_gpu, kernelRows,  kernelCols,
                              kernelStepY, kernelStepX,
                              numKernels,
        kernelBiases_gpu, scaleFactor,
        outputPtr_gpu
    );

    s_cudaCopyDeviceToHost(outputPtr, outputPtr_gpu, inputCount * outputStride);

    s_cudaFree(inputPtr_gpu);
    s_cudaFree(kernelPtr_gpu);
    s_cudaFree(kernelBiases_gpu);
    s_cudaFree(outputPtr_gpu);
}


void conv2d_backprop_multi_input_with_memcpy(
        u32 inputCount,  u32 inputStride,  u32 outputStride,
              fml* di_ptr,    u32 inputRows,   u32 inputCols,   u32 inputComponents,
        const fml* kernelPtr, u32 kernelRows,  u32 kernelCols,
                              u32 kernelStepY, u32 kernelStepX,
                              u32 numKernels,
        const fml* kernelBiases, fml scaleFactor,
        const fml* dA_ptr)
{
    fml* di_ptr_gpu = s_cudaMalloc(inputCount * inputStride);
    fml* kernelPtr_gpu = s_cudaMalloc(kernelRows * kernelCols * inputComponents * numKernels);
    fml* kernelBiases_gpu = s_cudaMalloc(numKernels);
    fml* dA_ptr_gpu = s_cudaMalloc(inputCount * outputStride);

    s_cudaCopyHostToDevice(kernelPtr_gpu, kernelPtr, kernelRows * kernelCols * inputComponents * numKernels);
    s_cudaCopyHostToDevice(kernelBiases_gpu, kernelBiases, numKernels);
    s_cudaCopyHostToDevice(dA_ptr_gpu, dA_ptr, inputCount * outputStride);

    conv2d_backprop_multi_input(
        inputCount,  inputStride,  outputStride,
        di_ptr_gpu,    inputRows,   inputCols,   inputComponents,
        kernelPtr_gpu, kernelRows,  kernelCols,
                              kernelStepY, kernelStepX,
                              numKernels,
        kernelBiases_gpu, scaleFactor,
        dA_ptr_gpu
    );

    s_cudaCopyDeviceToHost(di_ptr, di_ptr_gpu, inputCount * inputStride);

    s_cudaFree(di_ptr_gpu);
    s_cudaFree(kernelPtr_gpu);
    s_cudaFree(kernelBiases_gpu);
    s_cudaFree(dA_ptr_gpu);
}


void conv2d_accumError_multi_input_with_memcpy(
        u32 inputCount,  u32 inputStride,  u32 outputStride,
        const fml* inputPtr, u32 inputRows,   u32 inputCols,   u32 inputComponents,
              fml* dk_ptr,   u32 kernelRows,  u32 kernelCols,
                             u32 kernelStepY, u32 kernelStepX,
                             u32 numKernels,
              fml* db_ptr, fml scaleFactor,
        const fml* dA_ptr)
{
    fml* inputPtr_gpu = s_cudaMalloc(inputCount * inputStride);
    fml* dk_ptr_gpu = s_cudaMalloc(kernelRows * kernelCols * inputComponents * numKernels);
    fml* db_ptr_gpu = s_cudaMalloc(numKernels);
    fml* dA_ptr_gpu = s_cudaMalloc(inputCount * outputStride);

    s_cudaCopyHostToDevice(inputPtr_gpu, inputPtr, inputCount * inputStride);
    s_cudaCopyHostToDevice(dA_ptr_gpu, dA_ptr, inputCount * outputStride);

    conv2d_accumError_multi_input(
        inputCount,  inputStride,  outputStride,
        inputPtr_gpu, inputRows,   inputCols,   inputComponents,
        dk_ptr_gpu,   kernelRows,  kernelCols,
                             kernelStepY, kernelStepX,
                             numKernels,
        db_ptr_gpu, scaleFactor,
        dA_ptr_gpu
    );

    s_cudaCopyDeviceToHost(dk_ptr, dk_ptr_gpu, kernelRows * kernelCols * inputComponents * numKernels);
    s_cudaCopyDeviceToHost(db_ptr, db_ptr_gpu, numKernels);

    s_cudaFree(inputPtr_gpu);
    s_cudaFree(dk_ptr_gpu);
    s_cudaFree(db_ptr_gpu);
    s_cudaFree(dA_ptr_gpu);
}


}  // namespace gpu
}  // namespace conv2d
}  // namespace ml
