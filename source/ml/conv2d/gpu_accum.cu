#include <ml/conv2d/gpu.h>

#include "gpu_accum.ipp"


namespace ml
{
namespace conv2d
{
namespace gpu
{


void conv2d_accumError_multi_input(
        u32 inputCount,  u32 inputStride,  u32 outputStride,
        const fml* inputPtr, u32 inputRows,   u32 inputCols,   u32 inputComponents,
              fml* dk_ptr,   u32 kernelRows,  u32 kernelCols,
                             u32 kernelStepY, u32 kernelStepX,
                             u32 numKernels,
              fml* db_ptr, fml scaleFactor,
        const fml* dA_ptr)
{
    assert(inputPtr && inputRows > 0 && inputCols > 0 && inputComponents > 0);
    assert(dk_ptr && (kernelRows % 2) == 1 && (kernelCols % 2) == 1);
    assert(kernelStepY > 0 && kernelStepX > 0 && numKernels > 0);
    assert(db_ptr);
    assert(dA_ptr);

    if (kernelRows != 1 && kernelRows != 3 && kernelRows != 5 && kernelRows != 7)
        throw eInvalidArgument("Unsupported kernelRows: must be 1, 3, 5, or 7.");
    if (kernelCols != 1 && kernelCols != 3 && kernelCols != 5 && kernelCols != 7)
        throw eInvalidArgument("Unsupported kernelCols: must be 1, 3, 5, or 7.");

    if (kernelStepY != 1)
        throw eInvalidArgument("Unsupported kernelStepY: must be in 1.");
    if (kernelStepX != 1)
        throw eInvalidArgument("Unsupported kernelStepX: must be in 1.");

    if (numKernels > MAX_KERNELS_SUPPORTED)
        throw eInvalidArgument("Unsupported numKernels: you specified too many!");

    u32 kernelRadiusY = kernelRows / 2;
    u32 kernelRadiusX = kernelCols / 2;

    u32 effectiveBlockSizeY = BLOCK_SIZE_Y - 2*kernelRadiusY;  // Each block of threads will fill
    u32 effectiveBlockSizeX = BLOCK_SIZE_X - 2*kernelRadiusX;  // a smaller block of output, because we need
                                                               // an "apron" so that our kernel doesn't fall off
                                                               // the side and into no-where-land.

    dim3 gridSize;
    gridSize.x = (inputCols-1) / effectiveBlockSizeX + 1;
    gridSize.y = (inputRows-1) / effectiveBlockSizeY + 1;
    gridSize.z = inputCount;

    dim3 blockSize;
    blockSize.x = BLOCK_SIZE_X;
    blockSize.y = BLOCK_SIZE_Y;
    blockSize.z = 1;

    bool canUseFastImpl = true;
    u32 sharedMemNeeded = (BLOCK_SIZE_Y*BLOCK_SIZE_X*inputComponents + BLOCK_SIZE_Y*BLOCK_SIZE_X) * sizeof(fml);
    if (sharedMemNeeded * DESIRED_BLOCKS_PER_SM > SHARED_MEM_AVAIL_PER_SM)
    {
        canUseFastImpl = false;
        sharedMemNeeded = (BLOCK_SIZE_Y*BLOCK_SIZE_X) * sizeof(fml);
    }

    thrust::device_ptr<fml> dk(dk_ptr);
    thrust::device_ptr<fml> db(db_ptr);
    thrust::fill(dk, dk+kernelRows*kernelCols*inputComponents*numKernels, FML(0.0));
    thrust::fill(db, db+numKernels,                                       FML(0.0));

    RUN_CONV2D_ACCUM_GPU_FUNTION

    hipError_t errSync  = hipGetLastError();
    if (errSync != hipSuccess)
        throw eRuntimeError(std::string("CUDA launch error: ") + hipGetErrorString(errSync));
}


}  // namespace gpu
}  // namespace conv2d
}  // namespace ml
