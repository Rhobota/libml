#include "hip/hip_runtime.h"
#include <ml2/tAnnLayerGPU.h>


namespace ml2
{


class tExpFunc
{
    public:

        fml operator()(fml val) const { return std::min(std::exp(val), FML(1e30)); }
};


class tLogisticFunc
{
    public:

        fml operator()(fml val) const { return logistic_function(val); }
};


class tDirLogisticFunc
{
    public:

        fml operator()(fml val) const { return derivative_of_logistic_function(val); }
};


class tHyperbolicFunc
{
    public:

        fml operator()(fml val) const { return hyperbolic_function(val); }
};


class tDirHyperbolicFunc
{
    public:

        fml operator()(fml val) const { return derivative_of_hyperbolic_function(val); }
};


class t_RMSPROP_update
{
    public:

        fml operator()(fml accum, fml accum_avg) const
        {
            return (accum_avg > FML(0.0)) ? (accum / std::sqrt(accum_avg)) : FML(0.0);
        }
};


tAnnLayerGPU::tAnnLayerGPU(nAnnLayerType type, nAnnLayerWeightUpdateRule rule,
                           u32 numInputDims, u32 numNeurons, algo::iLCG& lcg,
                           fml randWeightMin, fml randWeightMax)
    : tAnnLayerBase(type, rule, numInputDims, numNeurons, lcg,
                    randWeightMin, randWeightMax)
{
}


tAnnLayerGPU::tAnnLayerGPU(iReadable* in)
    : tAnnLayerBase(in)
{
}


tAnnLayerGPU::~tAnnLayerGPU()
{
    // The super d'tor are called automatically.
}


void tAnnLayerGPU::takeInput(const fml* input, u32 numInputDims, u32 count)
{
    // TODO
}


const fml* tAnnLayerGPU::getOutput(u32& numOutputDims, u32& count) const
{
    // TODO
    return NULL;
}


void tAnnLayerGPU::takeOutputErrorGradients(
                  const fml* outputErrorGradients, u32 numOutputDims, u32 outputCount,
                  const fml* input, u32 numInputDims, u32 inputCount,
                  bool calculateInputErrorGradients)
{
    // TODO
}


const fml* tAnnLayerGPU::getInputErrorGradients(u32& numInputDims, u32& count) const
{
    // TODO
    return NULL;
}


static
iLayer* s_newLayerFunc(iReadable* in)
{
    return new tAnnLayerGPU(in);
}


static u32 layerId = 78879;
static bool didRegister = iLayer::registerLayerFuncWithHeaderId(s_newLayerFunc, layerId);


u32 tAnnLayerGPU::headerId() const
{
    if (!didRegister)
        throw eRuntimeError("Registering my layer id didn't work!");
    return layerId;
}


}   // namespace ml2
